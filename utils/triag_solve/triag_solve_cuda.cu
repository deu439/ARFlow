#include "hip/hip_runtime.h"
#include <torch/extension.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>


template <typename scalar_t>
__global__ void forward_substitution_kernel(
    const torch::PackedTensorAccessor32<scalar_t, 4, torch::RestrictPtrTraits> A,
    const torch::PackedTensorAccessor32<scalar_t, 4, torch::RestrictPtrTraits> B,
    const torch::PackedTensorAccessor32<scalar_t, 4, torch::RestrictPtrTraits> C,
    torch::PackedTensorAccessor32<scalar_t, 4, torch::RestrictPtrTraits> Y) {

    int M = A.size(2);
    int N = A.size(3);

    // Get the index of thread
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int i = index / A.size(1);  // batch
    int j = index % A.size(1);  // channel

    if (i < A.size(0) && j < A.size(1)) {
        for (int k = 0; k < M; k++) {
            for (int l = 0; l < N; l++) {
                if (k > 0)
                    Y[i][j][k][l] = Y[i][j][k][l] -  Y[i][j][k - 1][l] * C[i][j][k - 1][l];
                if (l > 0)
                    Y[i][j][k][l] = Y[i][j][k][l] - Y[i][j][k][l - 1] * B[i][j][k][l - 1];

                Y[i][j][k][l] = Y[i][j][k][l] / A[i][j][k][l];
            }
        }
    }
}

template <typename scalar_t>
__global__ void backward_substitution_kernel(
    const torch::PackedTensorAccessor32<scalar_t, 4, torch::RestrictPtrTraits> A,
    const torch::PackedTensorAccessor32<scalar_t, 4, torch::RestrictPtrTraits> B,
    const torch::PackedTensorAccessor32<scalar_t, 4, torch::RestrictPtrTraits> C,
    torch::PackedTensorAccessor32<scalar_t, 4, torch::RestrictPtrTraits> Y) {

    int M = A.size(2);
    int N = A.size(3);

    // Get the index of thread
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int i = index / A.size(1);  // batch
    int j = index % A.size(1);  // channel

    if (i < A.size(0) && j < A.size(1)) {
        for (int k = M-1; k >= 0; k--) {        // y axis
            for (int l = N-1; l >= 0; l--) {    // x axis
                if (k < M-1)
                    Y[i][j][k][l] = Y[i][j][k][l] -  Y[i][j][k + 1][l] * C[i][j][k][l];
                if (l < N-1)
                    Y[i][j][k][l] = Y[i][j][k][l] - Y[i][j][k][l + 1] * B[i][j][k][l];

                Y[i][j][k][l] = Y[i][j][k][l] / A[i][j][k][l];
            }
        }
    }
}


template <typename scalar_t>
__global__ void inverse_diagonal_kernel(
    const torch::PackedTensorAccessor32<scalar_t, 4, torch::RestrictPtrTraits> A,
    const torch::PackedTensorAccessor32<scalar_t, 4, torch::RestrictPtrTraits> B,
    const torch::PackedTensorAccessor32<scalar_t, 4, torch::RestrictPtrTraits> C,
    torch::PackedTensorAccessor32<scalar_t, 4, torch::RestrictPtrTraits> Y,
    torch::PackedTensorAccessor32<scalar_t, 4, torch::RestrictPtrTraits> H) {

    int M = A.size(2);
    int N = A.size(3);

    // Get the index of thread
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int m = index / A.size(1);  // batch
    int n = index % A.size(1);  // channel

    scalar_t sum;
    //scalar_t c, t, y;

    if (m < A.size(0) && n < A.size(1)) {
        for (int k = 0; k < M; k++){
            for (int l = 0; l < N; l++){
                sum = 0.0;
                //c = 0.0;

                // For i=k iterate over j=l,...,N
                for (int j = l; j < N; j++){
                    if (j == l)
                        Y[m][n][k][j] = 1.0;
                    if (j > l)
                        Y[m][n][k][j] = -Y[m][n][k][j - 1] * B[m][n][k][j - 1];

                    Y[m][n][k][j] = Y[m][n][k][j] / A[m][n][k][j];
                    //sum = sum + pow(Y[m][n][k][j], 2);
                    sum = sum + Y[m][n][k][j]*Y[m][n][k][j];
                    // Kahan summation
                    //y = pow(Y[m][n][k][j], 2) - c;
                    //t = sum + y;
                    //c = (t - sum) - y;
                    //sum = t;
                }

                // For i=k+1,...,M iterate over j=0,...,N
                for (int i = k+1; i < M; i++) {
                    for (int j = 0; j < N; j++) {
                        Y[m][n][i][j] = 0.0;

                        if (j >= l || i > k+1)  // If i=k+1 only upper neighbors j=l,...,N are valid
                            Y[m][n][i][j] = Y[m][n][i][j] -  Y[m][n][i - 1][j] * C[m][n][i - 1][j];
                        if (j > 0)
                            Y[m][n][i][j] = Y[m][n][i][j] - Y[m][n][i][j - 1] * B[m][n][i][j - 1];

                        Y[m][n][i][j] = Y[m][n][i][j] / A[m][n][i][j];
                        //sum = sum + pow(Y[m][n][i][j], 2);
                        sum = sum + Y[m][n][i][j]*Y[m][n][i][j];
                        // Kahan summation
                        //y = pow(Y[m][n][i][j], 2) - c;
                        //t = sum + y;
                        //c = (t - sum) - y;
                        //sum = t;
                    }
                }

                H[m][n][k][l] = sum;
            }
        }
    }
}

torch::Tensor forward_substitution_cuda(torch::Tensor A, torch::Tensor B, torch::Tensor C, torch::Tensor Y) {
    // Compute required number of blocks and threads
    int block_size = 32;
    int num_blocks = ceil((A.size(0) * A.size(1)) / (float)block_size);

    // Call corresponding CUDA kernel
    AT_DISPATCH_FLOATING_TYPES(A.type(), "forward_substitution_kernel", ([&] {
        forward_substitution_kernel<scalar_t><<<num_blocks, block_size>>>(
            A.packed_accessor32<scalar_t, 4, torch::RestrictPtrTraits>(),
            B.packed_accessor32<scalar_t, 4, torch::RestrictPtrTraits>(),
            C.packed_accessor32<scalar_t, 4, torch::RestrictPtrTraits>(),
            Y.packed_accessor32<scalar_t, 4, torch::RestrictPtrTraits>());
    }));

    return Y;
}

torch::Tensor backward_substitution_cuda(torch::Tensor A, torch::Tensor B, torch::Tensor C, torch::Tensor Y) {
    // Compute required number of blocks and threads
    int block_size = 32;
    int num_blocks = ceil((A.size(0) * A.size(1)) / (float)block_size);

    // Call corresponding CUDA kernel
    AT_DISPATCH_FLOATING_TYPES(A.type(), "backward_substitution_kernel", ([&] {
        backward_substitution_kernel<scalar_t><<<num_blocks, block_size>>>(
            A.packed_accessor32<scalar_t, 4, torch::RestrictPtrTraits>(),
            B.packed_accessor32<scalar_t, 4, torch::RestrictPtrTraits>(),
            C.packed_accessor32<scalar_t, 4, torch::RestrictPtrTraits>(),
            Y.packed_accessor32<scalar_t, 4, torch::RestrictPtrTraits>());
    }));

    return Y;
}

torch::Tensor inverse_diagonal_cuda(torch::Tensor A, torch::Tensor B, torch::Tensor C) {
    // Output & auxiliary tensors
    torch::Tensor H = torch::zeros_like(A);
    torch::Tensor Y = torch::zeros_like(A);

    // Compute required number of blocks and threads
    int block_size = 32;
    int num_blocks = ceil((A.size(0) * A.size(1)) / (float)block_size);

    // Call corresponding CUDA kernel
    AT_DISPATCH_FLOATING_TYPES(A.type(), "inverse_diagonal_kernel", ([&] {
        inverse_diagonal_kernel<scalar_t><<<num_blocks, block_size>>>(
            A.packed_accessor32<scalar_t, 4, torch::RestrictPtrTraits>(),
            B.packed_accessor32<scalar_t, 4, torch::RestrictPtrTraits>(),
            C.packed_accessor32<scalar_t, 4, torch::RestrictPtrTraits>(),
            Y.packed_accessor32<scalar_t, 4, torch::RestrictPtrTraits>(),
            H.packed_accessor32<scalar_t, 4, torch::RestrictPtrTraits>());
    }));

    return H;
}
